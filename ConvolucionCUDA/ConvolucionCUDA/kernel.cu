#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "mypgm.h"

#define SIZE 4096
#define MAXN 4096*4096
#define N 16384
#define L 1024
#define INF (1<<30)
#define MIN_(a,b) (((a)<(b))?(a):(b))
#define MAX_(a,b) (((a)>(b))?(a):(b))
#define sqr(x) (x)*(x)

typedef float uc;
float minimo = INF;
float maximo = -INF;

__global__ void kernel(uc *img1, uc *img2, float *mask, int *radio){
	int offset = (blockIdx.x * L) + threadIdx.x;
	float p, m;
	float suma = 0;
	int radio2;
	
	p = 0.0;
	int idx = offset, cont = 0;

	radio2 = *radio / 2;
	suma = 0;
	int q = 0;

	for (int h = -radio2; h <= radio2; h++)
	{
		for (int r = 0; r < *radio; r++)
		{
			p = img1[idx + SIZE*h + r - 1];
			m = mask[q]; 
			q++;
			suma += p*m;
		}
	}

	img2[idx] = suma;
}

uc* vectorize(){
	uc *img = new uc[SIZE*SIZE];
	for (size_t i = 0; i < SIZE; i++)
		for (size_t j = 0; j < SIZE; j++){
			size_t pos = (i*SIZE) + j;
			img[pos] = image1[i][j];
		}
	return img;
}

int* values(int r){
	int *img = new int[r*r];
	for (size_t i = 0; i < r; i++)
		for (size_t j = 0; j < r; j++){
			size_t pos = (i*r) + j;
			img[pos] = 1;
		}
	return img;
}

void vector_to_matrix(uc* img){
	for (size_t i = 0; i < SIZE; i++)
		for (size_t j = 0; j < SIZE; j++){
			size_t pos = SIZE*i + j;
			image2[i][j] = img[pos];
		}
}


int main(void)
{
	load_image_data();
	uc *dev_img1;
	uc *dev_img2;
	float *dev_mask;

	int r = 21, *dev_r;
	uc *img1 = vectorize();
	uc *img2 = new uc[SIZE*SIZE];
	float *mask = new float[r*r];

	for (auto i = 0; i < r*r; i++)
		mask[i] = (float)1.0 / (float)(r*r);



	hipMalloc((void**)&dev_img1, SIZE*SIZE*sizeof(float));
	hipMalloc((void**)&dev_img2, SIZE*SIZE*sizeof(float));
	hipMalloc((void**)&dev_mask, r*r*sizeof(float));
	hipMalloc((void**)&dev_r, sizeof(int));

	hipMemcpy(dev_img1, img1, SIZE*SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_mask, mask, r * r * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_r, &r, sizeof(int), hipMemcpyHostToDevice);

	float ms;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Haciendo el calculo de tiempo en el device
	kernel << <N, L >> >(dev_img1, dev_img2, dev_mask, dev_r);
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("GPUassert: %s\n", hipGetErrorString(hipGetLastError()));


	double memXFers = 5 * 4 * 4096 * 4096;
	memXFers /= 1024 * 1024 * 1024;
	printf("GPU: %f milisegundos con un ancho de banda %g GB/s\n", ms, memXFers / (ms / 1000.0));
	printf("Segundos: %f\n",ms/1000);

	hipMemcpy(img2, dev_img2, SIZE*SIZE*sizeof(float), hipMemcpyDeviceToHost);

	vector_to_matrix(img2);

	x_size2 = SIZE;
	y_size2 = SIZE;

	save_image_data();

	return 0;
}